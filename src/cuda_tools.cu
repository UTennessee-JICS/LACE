/*
    -- LACE (version 0.0) --
       Univ. of Tennessee, Knoxville

       @author Chad Burdyshaw
*/
#include "../include/sparse.h"
#include "../include/cuda_tools.h"

//#include "hipblas.h"
//#include "hipblas.h"
//#include "hipsparse.h"
//#include "hipsparse.h"

#if 0 //defined in hipblas.h
const char* hipGetErrorString(hipError_t status)
{
  char* status_string;
  if(status != hipSuccess)
  {
    switch (status)
    {
        case hipErrorInvalidValue:
            status_string = "hipErrorInvalidValue";
            break;
        case hipErrorOutOfMemory:
            status_string = "hipErrorOutOfMemory";
            break;
        case hipErrorInvalidDevicePointer:
            status_string = "hipErrorInvalidDevicePointer";
            break;
        default:
            status_string = "<unknown>";
    }
  }
  return status_string;
}
#endif

const char* cublasGetErrorString(hipblasStatus_t status)
{
  char* status_string;
  if(status != HIPBLAS_STATUS_SUCCESS)
  {
    switch (status)
    {
        case HIPBLAS_STATUS_SUCCESS:
            status_string = "HIPBLAS_STATUS_SUCCESS";
            break;
        case HIPBLAS_STATUS_NOT_INITIALIZED:
            status_string = "HIPBLAS_STATUS_NOT_INITIALIZED";
            break;
        case HIPBLAS_STATUS_ALLOC_FAILED:
            status_string = "HIPBLAS_STATUS_ALLOC_FAILED";
            break;
        case HIPBLAS_STATUS_INVALID_VALUE:
            status_string = "HIPBLAS_STATUS_INVALID_VALUE";
            break;
        case HIPBLAS_STATUS_ARCH_MISMATCH:
            status_string = "HIPBLAS_STATUS_ARCH_MISMATCH";
            break;
        case HIPBLAS_STATUS_MAPPING_ERROR:
            status_string = "HIPBLAS_STATUS_MAPPING_ERROR";
            break;
        case HIPBLAS_STATUS_EXECUTION_FAILED:
            status_string = "HIPBLAS_STATUS_EXECUTION_FAILED";
            break;
        case HIPBLAS_STATUS_INTERNAL_ERROR:
            status_string = "HIPBLAS_STATUS_INTERNAL_ERROR";
            break;
        default:
            status_string = "<unknown>";
    }
  }
  return status_string;
}


const char* hipsparseGetErrorString(hipsparseStatus_t status)
{
  char* status_string;
  if(status != HIPSPARSE_STATUS_SUCCESS)
  {
    switch (status)
    {
        case HIPSPARSE_STATUS_SUCCESS:
            status_string = "HIPSPARSE_STATUS_SUCCESS";
            break;
        case HIPSPARSE_STATUS_NOT_INITIALIZED:
            status_string = "HIPSPARSE_STATUS_NOT_INITIALIZED";
            break;
        case HIPSPARSE_STATUS_ALLOC_FAILED:
            status_string = "HIPSPARSE_STATUS_ALLOC_FAILED";
            break;
        case HIPSPARSE_STATUS_INVALID_VALUE:
            status_string = "HIPSPARSE_STATUS_INVALID_VALUE";
            break;
        case HIPSPARSE_STATUS_ARCH_MISMATCH:
            status_string = "HIPSPARSE_STATUS_ARCH_MISMATCH";
            break;
        case HIPSPARSE_STATUS_MAPPING_ERROR:
            status_string = "HIPSPARSE_STATUS_MAPPING_ERROR";
            break;
        case HIPSPARSE_STATUS_EXECUTION_FAILED:
            status_string = "HIPSPARSE_STATUS_EXECUTION_FAILED";
            break;
        case HIPSPARSE_STATUS_INTERNAL_ERROR:
            status_string = "HIPSPARSE_STATUS_INTERNAL_ERROR";
            break;
        case HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
            status_string = "HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED";
            break;
        default:
            status_string = "<unknown>";
    }
  }
  return status_string;
}


void cudaCheckCore(hipError_t code, const char* file, int line) {
   if (code != hipSuccess) {
      fprintf(stderr,"Cuda Error %d : %s %s %d\n", code, hipGetErrorString(code), file, line);
      exit(code);
   }
}
void cublasCheckCore(hipblasStatus_t code, const char* file, int line) {
   if (code != HIPBLAS_STATUS_SUCCESS) {
      fprintf(stderr,"CuBlas Error %d : %s %s %d\n", code, cublasGetErrorString(code), file, line);
      exit(code);
   }
}

void cusparseCheckCore(hipsparseStatus_t code, const char* file, int line) {
   if (code != HIPSPARSE_STATUS_SUCCESS) {
      fprintf(stderr,"CuSparse Error %d : %s %s %d\n", code, hipsparseGetErrorString(code), file, line);
      exit(code);
   }
}

